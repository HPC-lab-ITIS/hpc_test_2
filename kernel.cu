#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void kernel_sum(const float* A, const float* B, float* C, int n_el);

void sum(const float* A, const float* B, float* C, int n_el) {

  int threadsPerBlock,blocksPerGrid;

  if (n_el<512){
    threadsPerBlock = n_el;
    blocksPerGrid   = 1;
  } else {
    threadsPerBlock = 512;
    blocksPerGrid   = ceil(double(n_el)/double(threadsPerBlock));
  }

  kernel_sum<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, n_el);
}

__global__ void kernel_sum(const float* A, const float* B, float* C, int n_el)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < n_el) C[tid] = A[tid] + B[tid];
}